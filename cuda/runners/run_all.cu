
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/wait.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <unistd.h>
#include <fcntl.h>
#include <string.h>


void run_task(int data_fd, const char* task, int argc) {
	int N_arr[6] = {10, 14, 18, 22, 25, 28};
	int heights[3] = {10, 14, 18};
	int widths[2] = {8, 12};
	for (int i = 0; i < 3; ++i) {
		for (int j = 0; j < 2; ++j) {
			int fds[2];
			pipe(fds);

			pid_t pid = fork();
			if (pid == 0) {
				close(fds[0]);
				dup2(fds[1], 1);
				close(fds[1]);
				if (argc == 1) {
					char N[20];
                        		sprintf(N, "%d", N_arr[i * 2 + j]);
					execlp(task, task, N, NULL);
				} else if (argc == 2) {
					char height[20];
					sprintf(height, "%d", heights[i]);
					char width[20];
					sprintf(width, "%d", widths[j]);
					execlp(task, task, height, width, NULL);
				}
			} else {
				close(fds[1]);
				wait(NULL);
				dup2(fds[0], 0);
				close(fds[0]);
	
				const int size = 100;
				char output[size];
				read(0, output, size);
	
				char* ptr_colon = strstr(output, ":");
				char* ptr_endl = strstr(ptr_colon, "max");
				
				int time_string_length = (int) (ptr_endl - ptr_colon - 3);
				char time_string[40];
				strncpy(time_string, ptr_colon + 2, time_string_length);
				time_string[time_string_length] = '\0';
	
				if (argc == 1) {
					dprintf(data_fd, "size %d time %s\n", N_arr[i * 2 + j], time_string);
					printf("task %s, N %d done\n", task, N_arr[i * 2 + j]);
				} else if (argc == 2) {
					dprintf(data_fd, "size %d time %s\n", heights[i] * widths[j], time_string);
					printf("task %s, height %d width %d done\n", task, heights[i],  widths[j]);
				}
			}
		}
	}
	dprintf(data_fd, "\n");
}

int main() {
	int data_fd = open("../build/data.txt", O_CREAT | O_WRONLY, 0666);
	ftruncate(data_fd, 0);
	run_task(data_fd, "./01-add", 1);
	run_task(data_fd, "./02-mul", 1);
	run_task(data_fd, "./03-matrix-add", 2);
	run_task(data_fd, "./04-matrix-vector-mul", 2);
	run_task(data_fd, "./05-scalar-mul", 1);
	run_task(data_fd, "./06-cosine-vector", 1);
	close(data_fd);
	return 0;
}
