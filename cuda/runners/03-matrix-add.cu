#include "hip/hip_runtime.h"
#include "KernelMatrixAdd.cuh"
#include <iostream>

#define BLOCK_SIZE 256

void FillMatrix(float* matrix, int height, int width) {
	for (int i = 0; i < height; ++i) {
		for (int j = 0; j < width; ++j) {
			if (i == j) {
				matrix[i * width + j] = 1;
			} else {
				matrix[i * width + j] = 0;
			}
		}
	}
}


int main(int argc, char* argv[]) {
	int height = 1 << strtol(argv[1], NULL, 10);
	int width = 1 << strtol(argv[2], NULL, 10);
	int matrix_size = height * width;
	int bytes_count = matrix_size * sizeof(float);
	float* h_A = new float[matrix_size];
	float* h_B = new float[matrix_size];
	float* h_C = new float[matrix_size];
	FillMatrix(h_A, height, width);
	FillMatrix(h_B, height, width);
	float* d_A;
	float* d_B;
	float* d_C;

	hipMalloc(&d_A, bytes_count);
	hipMalloc(&d_B, bytes_count);
	hipMalloc(&d_C, bytes_count);
	hipMemcpy(d_A, h_A, bytes_count, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, bytes_count, hipMemcpyHostToDevice);

	int num_blocks = (height * width + BLOCK_SIZE - 1) / BLOCK_SIZE;

	hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);
	
	KernelMatrixAdd<<<num_blocks, BLOCK_SIZE>>>(height, width, 0, d_A, d_B, d_C);
	
	hipDeviceSynchronize();
	hipEventRecord(stop);
        float milliseconds;
        hipEventElapsedTime(&milliseconds, start, stop);
        std::cout << "time elapsed: " << milliseconds << std::endl;
	hipMemcpy(h_C, d_C, bytes_count, hipMemcpyDeviceToHost);
	
	float max_error = 0.0f;
	for (int i = 0; i < height; ++i) {
		for (int j = 0; j < width; ++j) {
			max_error = fmax(max_error, fabs(h_C[i * width + j] - (i == j ? 2.0f : 0.0f)));
		}
	}
	std::cout << "max error: " << max_error << std::endl;
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	delete[] h_A;
	delete[] h_B;
	delete[] h_C;

	return 0;
}
