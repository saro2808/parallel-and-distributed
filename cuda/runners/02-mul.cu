#include "hip/hip_runtime.h"
#include <KernelMul.cuh>
#include <iostream>

#define BLOCK_SIZE 256

int main(int argc, char* argv[]) {
	int N = 1 << strtol(argv[1], NULL, 10);
	size_t size = N * sizeof(float);
	float *h_x = (float*)malloc(size);
	float *h_y = (float*)malloc(size);
	float *h_z = (float*)malloc(size);

	float *d_x, *d_y, *d_z;

	hipMalloc(&d_x, size);
	hipMalloc(&d_y, size);
	hipMalloc(&d_z, size);

	for (int i = 0; i < N; ++i) {
		h_x[i] = 1.0f;
		h_y[i] = 2.0f;
	}


	hipMemcpy(d_x, h_x, size, hipMemcpyHostToDevice);
	hipMemcpy(d_y, h_y, size, hipMemcpyHostToDevice);

	int numBlocks = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;

	hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);
	
	KernelMul<<<numBlocks/ILP, BLOCK_SIZE>>>(N, d_x, d_y, d_z);

	hipDeviceSynchronize();	
	hipEventRecord(stop);
        float milliseconds;
        hipEventElapsedTime(&milliseconds, start, stop);
        std::cout << "time elapsed: " << milliseconds << std::endl;

	hipMemcpy(h_z, d_z, size, hipMemcpyDeviceToHost);

	float maxError = 0.0f;
	for (int i = 0; i < N; ++i) {
		maxError = fmax(maxError, fabs(h_z[i]-2.0f));
	}
	std::cout << "max error: " << maxError << std::endl;

	hipFree(d_x);
	hipFree(d_y);
	hipFree(d_z);
	free(h_x);
	free(h_y);
	free(h_z);
	return 0;
}
