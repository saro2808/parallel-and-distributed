#include "hip/hip_runtime.h"
#include <CosineVector.cuh>
#include <iostream>

#define BLOCK_SIZE 256

int main(int argc, char* argv[]) {
	int N = 1 << strtol(argv[1], NULL, 10);
	float* vector1 = new float[N];
	float* vector2 = new float[N];
	for (int i = 0; i < N; ++i) vector1[i] = vector2[i] = 1;
	hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);
        
	int cosine = CosineVector(N, vector1, vector2, BLOCK_SIZE);
	
	hipDeviceSynchronize();
	hipEventRecord(stop);
        float milliseconds;
        hipEventElapsedTime(&milliseconds, start, stop);
        std::cout << "time elapsed: " << milliseconds << std::endl;

        std::cout << "max error: " << fabs(cosine - 1) << std::endl;
	delete[] vector1;
	delete[] vector2;
}

