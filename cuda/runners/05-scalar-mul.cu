#include "hip/hip_runtime.h"
#include <ScalarMulRunner.cuh>
#include <iostream>

#define BLOCK_SIZE 256

int main(int argc, char* argv[]) {
	int N = 1 << strtol(argv[1], NULL, 10);
	float* vector1 = new float[N];
	float* vector2 = new float[N];
	for (int i = 0; i < N; ++i) {
		vector1[i] = 1.0f;
		vector2[i] = 1.0f;
	}
	float* d_vector1;
	float* d_vector2;
	hipMalloc(&d_vector1, N * sizeof(float));
	hipMalloc(&d_vector2, N * sizeof(float));
	hipMemcpy(d_vector1, vector1, N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_vector2, vector2, N * sizeof(float), hipMemcpyHostToDevice);

	hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);
	
	float res = ScalarMulSumPlusReduction(N, d_vector1, d_vector2, BLOCK_SIZE);

	hipDeviceSynchronize();
	hipEventRecord(stop);
        float milliseconds;
        hipEventElapsedTime(&milliseconds, start, stop);
        std::cout << "time elapsed: " << milliseconds << std::endl;

	std::cout << "max error: " << fabs(res - N) << std::endl;

	hipFree(d_vector1);
	hipFree(d_vector2);
	delete[] vector2;
	delete[] vector1;
	return 0;
}

