#include "hip/hip_runtime.h"
#include <MatrixVectorMul.cuh>
#include <iostream>

#define BLOCK_SIZE 256

int main(int argc, char* argv[]) {
	int height = 1 << strtol(argv[1], NULL, 10);
	int width = 1 << strtol(argv[2], NULL, 10);
	int matrix_size = height * width;
	int bytes_count = matrix_size * sizeof(float);
	
	float* matrix = new float[matrix_size];
	float* vector = new float[width];
	float* result = new float[height];
	
	for (int i = 0; i < height; ++i) {
		for (int j = 0; j < width; ++j) {
			matrix[i * width + j] = (i == j) ? 1 : 0;
		}
		result[i] = 0;
	}
	for (int j = 0; j < width; ++j) vector[j] = 1;
	
	float* d_matrix;
	float* d_vector;
	float* d_result;
	
	hipMalloc(&d_matrix, bytes_count);
	hipMalloc(&d_vector, width * sizeof(float));
	hipMalloc(&d_result, height * sizeof(float));
	
	hipMemcpy(d_matrix, matrix, bytes_count, hipMemcpyHostToDevice);
	hipMemcpy(d_vector, vector, width * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_result, result, height * sizeof(float), hipMemcpyHostToDevice);
	
	int blocksNum = (matrix_size + BLOCK_SIZE - 1) / BLOCK_SIZE;
	
	hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);
	
	MatrixVectorMul<<<blocksNum, BLOCK_SIZE>>>(height, width, d_matrix, d_vector, d_result);
	
	hipDeviceSynchronize();
        hipEventRecord(stop);
        float milliseconds;
        hipEventElapsedTime(&milliseconds, start, stop);
        std::cout << "time elapsed: " << milliseconds << std::endl;
	
	hipMemcpy(result, d_result, height * sizeof(float), hipMemcpyDeviceToHost);
	
	float max_error = 0.0f;
	for (int i = 0; i < height; ++i) {
		max_error = fmax(max_error, fabs(result[i] - (i < width ? 1.0f : 0.0f)));
	}
	std::cout << "max error: " << max_error << std::endl;

	hipFree(d_result);
	hipFree(d_matrix);
	hipFree(d_vector);
	delete[] result;
	delete[] vector;
	delete[] matrix;
}

