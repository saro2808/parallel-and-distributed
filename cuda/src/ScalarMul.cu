#include "hip/hip_runtime.h"
#include <ScalarMul.cuh>
#include <stdio.h>

/*
 * Calculates scalar multiplication for block
 */
__global__
void ScalarMulBlock(int numElements, float* vector1, float* vector2, float *result) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < numElements; i += stride) {
		atomicAdd(result + blockIdx.x, vector1[i] * vector2[i]);
	}
	/*extern __shared__ float shared_data[];
	
	int index = threadIdx.x + blockIdx.x * blockDim.x;
        int stride = blockDim.x * gridDim.x;
printf("cp1 ");
	if (threadIdx.x == 0) {
		printf("threadIdx.x == %d\n", threadIdx.x);
		shared_data[blockIdx.x] = 0;
		__syncthreads();
	}
	
	for (int i = index; i < numElements; i += stride) {
		//printf("cp1.5 ");
                atomicAdd(shared_data + blockIdx.x, vector1[i] * vector2[i]);
		//shared_data[blockIdx.x] += vector1[i] * vector2[i];
		printf("cp1.75 ");
		__syncthreads();
        }
printf("cp2 ");
	__syncthreads();
printf("cp3 ");
	if (threadIdx.x == 0) {
		result[blockIdx.x] = shared_data[blockIdx.x];
		printf("[%d]%f ", blockIdx.x, shared_data[blockIdx.x]);
	}
	printf("returning from kernel ");*/
}

