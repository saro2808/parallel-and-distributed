#include "hip/hip_runtime.h"
#include <KernelMul.cuh>

__global__ void KernelMul(int numElements, float* x, float* y, float* result) {
	int tid = threadIdx.x + ILP * blockDim.x * blockIdx.x;
	for (int i = 0; i < ILP; ++i) {
		int current_tid = tid + i * blockDim.x;
		result[current_tid] = x[current_tid] * y[current_tid];
	}
}

