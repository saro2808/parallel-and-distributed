#include "hip/hip_runtime.h"
#include "KernelAdd.cuh"

__global__ void KernelAdd(int numElements, float* x, float* y, float* result) {
	int tid = threadIdx.x + ILP * blockDim.x * blockIdx.x;
	for (int i = 0; i < ILP; ++i) {
		int current_tid = tid + i * blockDim.x;
		result[current_tid] = x[current_tid] + y[current_tid];
	}
}
