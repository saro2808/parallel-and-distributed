#include "hip/hip_runtime.h"
#include <MatrixVectorMul.cuh>

__global__
void MatrixVectorMul(int height, int width, float* matrix, float* vector, float* result) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int jndex = threadIdx.y + blockIdx.y * blockDim.y;
	int stride_i = blockDim.x * gridDim.x;
	int stride_j = blockDim.y * gridDim.y;
	for (int i = index; i < height; i += stride_i) {
		for (int j = jndex; j < width; j += stride_j) {
			result[i] += matrix[i * width + j] * vector[j];
		}
	}
}

