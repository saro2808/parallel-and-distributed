#include "hip/hip_runtime.h"
#include <ScalarMulRunner.cuh>
#include <ScalarMul.cuh>

float ScalarMulTwoReductions(int numElements, float* vector1, float* vector2, int blockSize) {
	
	return 0.0f;
}

__global__
void BlockSum(int blocks_count, float* blocks_vector, float* d_dot_product) {
	int index = threadIdx.x;
	for (int i = index; i < blocks_count; i += blockDim.x) {
		atomicAdd(d_dot_product, blocks_vector[i]);
	}
}

float ScalarMulSumPlusReduction(int numElements, float* d_vector1, float* d_vector2, int blockSize) {
	float dot_product = 0;
	int blocksCount = (numElements + blockSize - 1) / blockSize;
	float* h_result = new float[blocksCount];
	for (int i = 0; i < blocksCount; ++i) {
		h_result[i] = 0.0f;
	}

	float* d_result;
	hipMalloc(&d_result, blocksCount * sizeof(float));
	hipMemcpy(d_result, h_result, blocksCount * sizeof(float), hipMemcpyHostToDevice);

	ScalarMulBlock<<<blocksCount, blockSize>>>(numElements, d_vector1, d_vector2, d_result);
	hipDeviceSynchronize();
	
	float* d_dot_product;
	hipMalloc(&d_dot_product, sizeof(float));
	hipMemcpy(d_dot_product, &dot_product, sizeof(float), hipMemcpyHostToDevice);
	
	BlockSum<<<1, blockSize>>>(blocksCount, d_result, d_dot_product);
	hipDeviceSynchronize();

	hipMemcpy(&dot_product, d_dot_product, sizeof(float), hipMemcpyDeviceToHost);
	
	hipFree(d_dot_product);
	delete[] h_result;
	hipFree(d_result);
	return dot_product;
}

