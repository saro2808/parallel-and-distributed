#include "hip/hip_runtime.h"
#include <MatrixMul.cuh>

__global__
void MatrixMul(int heightA, int widthA, int widthB, float *matrixA, float *matrixB, float *matrixResult) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	//int height = blockDim.x * gridDim.x;
	//int width = blockDim.y * gridDim.y;

	matrixResult[i * widthB + j] = .0f;

	for (int k = 0; k < widthA; ++k) {
		matrixResult[i * widthB + j] += matrixA[i * widthA + k] * matrixB[k * widthB + j];
	}
}

