#include "hip/hip_runtime.h"
#include <CosineVector.cuh>
#include <ScalarMulRunner.cuh>
#include <cmath>
#include <iostream>

float CosineVector(int numElements, float* vector1, float* vector2, int blockSize) {
	float* d_vector1;
	float* d_vector2;
	int size = numElements * sizeof(float);
	hipMalloc(&d_vector1, size);
	hipMalloc(&d_vector2, size);
	hipMemcpy(d_vector1, vector1, size, hipMemcpyHostToDevice);
	hipMemcpy(d_vector2, vector2, size, hipMemcpyHostToDevice);
	float vector1_norm = sqrt(ScalarMulSumPlusReduction(numElements, d_vector1, d_vector1, blockSize));
	float vector2_norm = sqrt(ScalarMulSumPlusReduction(numElements, d_vector2, d_vector2, blockSize));
	float vector1_dot_vector2 = ScalarMulSumPlusReduction(numElements, d_vector1, d_vector2, blockSize);
	hipFree(d_vector1);
	hipFree(d_vector2);
	return vector1_dot_vector2 / vector1_norm / vector2_norm;
}

