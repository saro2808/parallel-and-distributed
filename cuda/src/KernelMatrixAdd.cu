#include "hip/hip_runtime.h"
#include <KernelMatrixAdd.cuh>
#include <stdio.h>

__global__ void KernelMatrixAdd(int height, int width, int pitch, float* A, float* B, float* result) {
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	int jndex = blockDim.y * blockIdx.y + threadIdx.y;
	int stride_i = blockDim.x * gridDim.x;
	int stride_j = blockDim.y * gridDim.y;
	for (int i = index; i < height; i += stride_i) {
		for (int j = jndex; j < width; j += stride_j) {
			result[i * width + j] = A[i * width + j] + B[i * width + j];
		}
	}
}

